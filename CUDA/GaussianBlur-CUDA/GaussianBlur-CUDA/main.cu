#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include "lodepng.h"

const int ker_x_dim = 3;
const int ker_y_dim = 3;
const double sigma = 1.0;

double *kernel = new double[(ker_x_dim * 2)*(ker_y_dim * 2)];

void getGaussianKernel() {
	// generate gaussian kernel values
	double temp, sum = 0.0;
	int r_i, r_j = 0;
	for (int i = -ker_x_dim; i <= ker_x_dim; i++) {
		r_i = i + ker_x_dim;
		for (int j = -ker_y_dim; j <= ker_y_dim; j++) {
			r_j = j + ker_y_dim;
			temp = exp(-((i*i) + (j*j)) / (2 * (sigma*sigma)));
			kernel[r_i*ker_y_dim+r_j] = temp / (2*M_PI*sigma*sigma);
			printf("[%d][%d] = %f, ", i, j, kernel[r_i*ker_y_dim+r_j]);
		}
	}
}

__global__ void runFilter(float *input, float *output) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = -ker_x_dim; i < ker_x_dim; ++i) {
		for (int j = -ker_y_dim; j < ker_y_dim; ++j) {

		}
	}
}

int main() {
	const char* image_path = "image.png";
	const char* output_path = "output.png";

	// create gaussian kernel 
	getGaussianKernel();

	std::vector<unsigned char> img_vect;
	unsigned int width, height;

	unsigned error = lodepng::decode(img_vect, width, height, image_path);
	if (error) {
		printf("decoder error: %d, %s", error, lodepng_error_text(error));
	}
	unsigned char* input = new unsigned char[(img_vect.size() * 3) / 4];
	unsigned char* output = new unsigned char[(img_vect.size() * 3) / 4];
	int count = 0;
	
	// getting rid of the apha channel as it is not needed
	for (int i = 0; i < img_vect.size(); ++i) {
		if ((i + 1) % 4 != 0) {
			input[count] = img_vect.at(i);
			output[count] = img_vect.at(i);
			count++;
		}
	}
	//printf("%d, %d, %d\n", input[0], input[1], input[2]);

	dim3 blockDim(16, 16, 1);
	dim3 gridDim(width / blockDim.x + 1, height / blockDim.y + 1);

	runFilter << <gridDim, blockDim >> > (input, output);
	

	// Prepare data for output
	/*std::vector<unsigned char> out_image;
	printf("test");
	for (int i = 0; i < img_vect.size(); ++i) {
		out_image.push_back(output[i]);
		if ((i + 1) % 3 == 0) {
			out_image.push_back(255);
		}
	}*/

	// Output the data
	//error = lodepng::encode(output_path, out_image, width, height);

	//if there's an error, display it
	//if (error) std::cout << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;

	
}

