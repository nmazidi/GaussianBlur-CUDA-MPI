#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <memory.h>
#include <time.h>
#include "lodepng.h"
#undef main

const int ker_x_dim = 3;
const int ker_y_dim = 3;
const double sigma = 1.0;
//double *kernel = new double[(ker_x_dim * 2)*(ker_y_dim * 2)];
float h_kernel[(ker_x_dim * 2)*(ker_y_dim * 2)];
__constant__ float d_kernel[(ker_x_dim * 2)*(ker_y_dim * 2)];

void getGaussianKernel() 
{
	// generate gaussian kernel values
	double temp = 0.0;
	int r_i, r_j = 0;
	for (int i = -ker_x_dim; i <= ker_x_dim; i++) {
		r_i = i + ker_x_dim;
		for (int j = -ker_y_dim; j <= ker_y_dim; j++) {
			r_j = j + ker_y_dim;
			temp = exp(-((i*i) + (j*j)) / (2 * (sigma*sigma)));
			h_kernel[r_i*ker_y_dim+r_j] = temp / (2*M_PI*sigma*sigma);
			printf("[%d][%d] = %f, ", i, j, h_kernel[r_i*ker_y_dim+r_j]);
		}
	}
}

__host__ __device__ int get1dIndex(int width, int height, int x, int y)
{
	if (x < 0) x = 0;
	if (x >= width) x = width - 1;
	if (y < 0) y = 0;
	if (y >= height) y = height - 1;
	return y*width + x;
}

__global__ void runFilter(float* input, float* output, int width, int height) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float new_val = 0.0f;
	int count = 0;
	int r_i, r_j = 0;
	if (row < width && col < height) {
		for (int i = -ker_x_dim; i < ker_x_dim; i++) {
			r_i = i + ker_x_dim;
			for (int j = -ker_y_dim; j < ker_y_dim; j++) {
				r_j = j + ker_y_dim;
				int idx = get1dIndex(width, height, row + i, col + j);

				new_val += d_kernel[r_i*ker_y_dim + r_j] * input[idx];
			}
		}
		output[get1dIndex(width, height, row, col)] = new_val;
	}
}
void convolveImage(float* input, float* output, int width, int height) 
{
	float* d_input;
	float* d_output;
	hipMalloc(&d_input, width*height * sizeof(float));
	hipMalloc(&d_output, width*height * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, sizeof(h_kernel));
	hipMemcpy(d_input, input, width*height * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockDim(25, 25, 1);
	dim3 gridDim(width / (blockDim.x) + 1, height / (blockDim.y) + 1);
	printf("height: %d, width: %d", height, width);

	runFilter << <gridDim, blockDim >> >(d_input, d_output, width, height);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));

	}
	hipDeviceSynchronize();
	hipMemcpy(output, d_output, width*height * sizeof(float), hipMemcpyDeviceToHost);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error running kernel: %s\n", hipGetErrorString(cudaStatus));
	}
}

int main(int argc, int** argv) 
{
	const char* image_path = "image.png";
	const char* output_path = "output.png";

	// create gaussian kernel 
	getGaussianKernel();

	std::vector<unsigned char> img_vect;
	unsigned int width, height;

	unsigned error = lodepng::decode(img_vect, width, height, image_path);
	if (error) {
		printf("decoder error: %d, %s", error, lodepng_error_text(error));
	}
	int image_size = width*height; 
	float* temp;
	float* input;
	float* output;
	hipHostMalloc(&temp, (image_size*3) * sizeof(float));
	hipHostMalloc(&input, (image_size) * sizeof(float));
	hipHostMalloc(&output, (image_size) * sizeof(float));
	int count = 0;
	// getting rid of the apha channel as it is not needed
	for (int i = 0; i < img_vect.size(); ++i) {
		if ((i + 1) % 4 != 0) {
			temp[count] = img_vect.at(i);
			count++;
		}
	}
	for (int i = 0; i < image_size; i++) {
		input[i] = (
			temp[i * 3 + 0] +
			temp[i * 3 + 1] +
			temp[i * 3 + 2])/3;
	}

	clock_t tStart = clock();
	convolveImage(input, output, width, height);
	clock_t tEnd = clock();
	float ms = 1000.0f * (tEnd - tStart) / CLOCKS_PER_SEC;
	printf("Convolution took %fms.\n", ms);
	//printf("%f * %f = %f", input[0], input[])
	printf("output: %f", output[0]);

	std::vector<unsigned char> out_image;
	for (int i = 0; i < image_size; i++) {
		out_image.push_back(output[i]);
		out_image.push_back(output[i]);
		out_image.push_back(output[i]);
		out_image.push_back(255);
		
	}
	error = lodepng::encode(output_path, out_image, width, height);

	//if there's an error, display it
	if (error) {
		printf("lodepng error: %s", lodepng_error_text(error));
	}
	
}

